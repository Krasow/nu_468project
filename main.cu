#include "hash.h"

#include <chrono>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

using namespace std::chrono;

// form https://www.google.com/search?client=firefox-b-1-d&q=store+file+into+array+in+c%2B%2B
void readFile(const char* filename, vector<string>& lines)
{
    lines.clear();
    ifstream file(filename);
    string s;
    while (getline(file, s))
        lines.push_back(s);
}

bool equal_passwords(string guess, string answer) {
    return (answer.compare(guess) == 0);
}


int main(){
    // stores the list of dictionary
    std::vector<string> dictionary; 
    readFile("wordlist.10000", dictionary);

    // stores the list of passwords wanting to crack
    std::vector<string> passwords;
    readFile("cracked.txt", passwords);

    auto start = high_resolution_clock::now();
    dict_cpu(passwords, dictionary);
    auto stop = high_resolution_clock::now();
    auto duration_dict_cpu = duration_cast<microseconds>(stop - start).count();

    start = high_resolution_clock::now();
    crack_cpu(passwords);
    stop = high_resolution_clock::now();
    auto duration_brute_force = duration_cast<microseconds>(stop - start).count();



    cout << duration_dict_cpu << duration_brute_force << endl;

    
    
    return 0;



}
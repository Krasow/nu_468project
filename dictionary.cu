#include "hip/hip_runtime.h"
#include "hash.h"

__global__ void dictionary_kernel(char* dictionary, char *passwords)
{
	// calculate the row index of the Pd element and M
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // calculate the column index of Pd and N
    int col = blockIdx.x * blockDim.x + threadIdx.x;

}
